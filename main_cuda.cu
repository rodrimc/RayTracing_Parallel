#include "hip/hip_runtime.h"
/*
 * main_cuda.cu
 *
 *  Created on: Oct 30, 2014
 *      Author: Rodrigo Costa
 *			e-mail: rodrigocosta@telemidia.puc-rio.br
 */

#include "c_util.h"

#include <math.h>
#include <stdio.h>
#include <string>
#include <sys/time.h>


int main (int argc, char** argv)
{
  int num_bytes, num_spheres, num_planes, num_lights;
  timeval t_start, t_end;
  double elapsed_time;
	Image *h_image, *d_image;
  Sphere *d_spheres;
  Plane *d_planes;
  Light *d_lights;
	std::string filename = "out.ppm";

  dim3 threadsPerBlock (16, 16);
  dim3 numBlocks;

	if (argc < 3)
	{
		printf ("Usage: %s <widht> <height> [<fov>]\n", argv[0]);
		return 0;
	}

	int width = atoi (argv[1]);;
	int height = atoi (argv[2]);;

  num_bytes = (width * height) * sizeof(Color);

	float fov = 60.0;
  if (argc >= 4)
  {
    fov = atof (argv[3]);
  }

  gettimeofday (&t_start, NULL);

  h_image = new Image[width * height];

  if (c_initScene (&d_spheres, &num_spheres, 
        &d_planes, &num_planes,
        &d_lights, &num_lights))
  {
    //Allocation of memory for the scene on device
    hipMalloc (&d_image, num_bytes);

    numBlocks = dim3 (width/threadsPerBlock.x + 1, height/threadsPerBlock.y + 1);

    float tanFov = tan (fov * 0.5 * M_PI / 180.0f);
    float aspect_ratio = float (width) / float (height);

    printf ("Rendering scene:\n");
    printf ("Width: %d \nHeight: %d\nFov: %.2f\n", width, height, fov);

    numBlocks = dim3 (width/threadsPerBlock.x + 1, height/threadsPerBlock.y + 1);

    printf ("Blocks: %d x %d\n", numBlocks.x, numBlocks.y);

    k_trace <<<numBlocks, threadsPerBlock>>> 
      (d_image, d_planes, num_planes, d_spheres, num_spheres, d_lights, 
       num_lights, aspect_ratio, tanFov, width, height);
    cudaCheckErrors ("Calling kernel k_test");

    gettimeofday (&t_end, NULL);

    elapsed_time = (t_end.tv_sec - t_start.tv_sec) * 1000.0;
    elapsed_time += (t_end.tv_usec - t_start.tv_usec) / 1000.0;

    printf ("\r100.00%%");
    printf ("\nFinished!\n");
    printf ("Rendering time: %.3f s\n", elapsed_time/1000.0);

    hipMemcpy (h_image, d_image, num_bytes, hipMemcpyDeviceToHost);
    writePPMFile (h_image, "output/cuda.ppm", width, height);
  }
  else
    printf ("ERROR. Exiting...\n");

  delete h_image;
  hipFree (d_image);
  hipFree (d_planes);
  hipFree (d_spheres);
  hipFree (d_lights);
  
	return 0;
}
